#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"

void sumArrays(float* a, float* b, float* res, const int size) {
    for (int i = 0; i < size; i += 4) {
        res[i]     = a[i] + b[i];
        res[i + 1] = a[i + 1] + b[i + 1];
        res[i + 2] = a[i + 2] + b[i + 2];
        res[i + 3] = a[i + 3] + b[i + 3];
    }
}

__global__ void sumArraysGPU(float* a, float* b, float* res) {
    int i  = blockIdx.x * blockDim.x + threadIdx.x;
    res[i] = a[i] + b[i];
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);

    int power = 14;
    if (argc >= 2)
        power = atoi(argv[1]);
    int nElem = 1 << power;
    printf("Vector size:%d\n", nElem);
    int    nByte          = sizeof(float) * nElem;
    float* res_from_gpu_h = (float*)malloc(nByte);
    float* res_h          = (float*)malloc(nByte);
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    float *a_host, *b_host, *res_d;
    double iStart, iElaps;
    dim3   block(1024);
    dim3   grid(nElem / block.x);
    res_from_gpu_h = (float*)malloc(nByte);
    CHECK(hipHostAlloc((float**)&a_host, nByte, hipHostMallocMapped));
    CHECK(hipHostAlloc((float**)&b_host, nByte, hipHostMallocMapped));
    CHECK(hipMalloc((float**)&res_d, nByte));
    initialData(a_host, nElem);
    initialData(b_host, nElem);

    //=============================================================//
    iStart = cpuSecond();
    sumArraysGPU<<<grid, block>>>(a_host, b_host, res_d);
    CHECK(hipMemcpy(res_from_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));
    iElaps = cpuSecond() - iStart;
    //=============================================================//
    printf("zero copy memory elapsed %lf ms \n", iElaps);
    printf("Execution configuration<<<%d,%d>>>\n", grid.x, block.x);

    //-----------------------CPU Memory--------------------------------
    sumArrays(a_host, b_host, res_h, nElem);
    checkResult(res_h, res_from_gpu_h, nElem);

    hipHostFree(a_host);
    hipHostFree(b_host);
    hipFree(res_d);
    free(res_h);
    free(res_from_gpu_h);

    return 0;
}