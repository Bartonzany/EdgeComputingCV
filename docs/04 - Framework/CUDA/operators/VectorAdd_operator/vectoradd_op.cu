
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int n = 1000; // 数组长度
    int *a, *b, *c; // 输入和输出数组

    // 为输入数组赋值
    a = (int*)malloc(n * sizeof(int));
    b = (int*)malloc(n * sizeof(int));
    c = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    int *dev_a, *dev_b, *dev_c; // 设备端指针

    // 分配设备端内存
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // 将输入数据从主机端拷贝到设备端
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // 计算线程块和线程数
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // 调用 CUDA 核函数
    vectorAdd<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // 将计算结果从设备端拷贝到主机端
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // 打印输出结果的部分内容
    printf("Result: ");
    for (int i = 0; i < 10; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // 释放主机端内存
    free(a);
    free(b);
    free(c);

    // 释放设备端内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
