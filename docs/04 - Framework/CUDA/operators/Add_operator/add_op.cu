
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addKernel(int *a, int *b, int *c, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  //当前线程的全局唯一标识符
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int n = 10; // 数组长度
    int a[n], b[n], c[n]; // 输入和输出数组

    // 为输入数组赋值
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    int *dev_a, *dev_b, *dev_c; // 设备端指针

    // 分配设备端内存
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // 在 GPU 上分配内存, 并将输入数据从主机端拷贝到设备端
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // 计算线程块和线程数，指定了线程块和线程的数量
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // 调用 CUDA 核函数
    addKernel<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // 将计算结果从设备端拷贝到主机端
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // 打印输出结果
    printf("Result: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // 释放设备端内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
