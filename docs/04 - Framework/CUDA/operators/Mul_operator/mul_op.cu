
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mulKernel(int *a, int *b, int *c, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        c[tid] = a[tid] * b[tid];
    }
}

int main() {
    int n = 10; // ���鳤��
    int a[n], b[n], c[n]; // ������������

    // Ϊ�������鸳ֵ
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    int *dev_a, *dev_b, *dev_c; // �豸��ָ��

    // �����豸���ڴ�
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // ���������ݴ������˿������豸��
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // �����߳̿���߳���
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // ���� CUDA �˺���
    mulKernel<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // �����������豸�˿�����������
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // ��ӡ������
    printf("Result: ");
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // �ͷ��豸���ڴ�
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
