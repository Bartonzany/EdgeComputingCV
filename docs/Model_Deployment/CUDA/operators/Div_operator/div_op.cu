
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void subtractKernel(float *a, float *b, float *c, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        if (b[tid] != 0) {
            c[tid] = a[tid] / b[tid];
        } else {
            c[tid] = 0; // �������Ϊ�㣬������Ϊ��������ʵ���ֵ
        }
    }
}

int main() {
    int n = 10; // ���鳤��
    float a[n], b[n], c[n]; // ������������

    // Ϊ�������鸳ֵ
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i * i;
    }

    float *dev_a, *dev_b, *dev_c; // �豸��ָ��

    // �����豸���ڴ�
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));
    hipMalloc((void**)&dev_c, n * sizeof(int));

    // ���������ݴ������˿������豸��
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // �����߳̿���߳���
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // ���� CUDA �˺���
    subtractKernel<<<numBlocks, blockSize>>>(dev_a, dev_b, dev_c, n);

    // �����������豸�˿�����������
    hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // ��ӡ������
    printf("Result: ");
    for (int i = 0; i < n; ++i) {
        printf("%f ", c[i]);
    }
    printf("\n");

    // �ͷ��豸���ڴ�
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
